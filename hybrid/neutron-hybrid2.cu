#include "hip/hip_runtime.h"
/*
 * Université Pierre et Marie Curie
 * Calcul de transport de neutrons
 * Version séquentielle
 */
//nvcc -o exec neutron-par.cu -O3 --generate-code arch =compute_35, code=sm_35 && ./exec 
//nvcc -o exec neutron-par.cu -O3 --generate-code arch=compute_35,code=sm_35 && ./exec

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>
#include<hiprand/hiprand_kernel.h>
#include <thrust/remove.h>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <omp.h>

#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#define THREADS_PER_BLOCK 1024
#define OUTPUT_FILE "/tmp/hybrid-absorbed.dat"

char info[] = "\
Usage:\n\
    neutron-seq H Nb C_c C_s\n\
\n\
    H  : épaisseur de la plaque\n\
    Nb : nombre d'échantillons\n\
    C_c: composante absorbante\n\
    C_s: componente diffusante\n\
\n\
Exemple d'execution : \n\
    neutron-seq 1.0 500000000 0.5 0.5\n\
";

/*
 * générateur uniforme de nombres aléatoires dans l'intervalle [0,1)
 */
struct drand48_data alea_buffer;


struct is_not_zero
{
  __host__ __device__
  bool operator()(float x)
  {
    return  x == 0;
  }
};


struct is_even
{
  __host__ __device__
  bool operator()(const int x)
  {
    return (x % 2) == 0;
  }
};


void init_uniform_random_number() {
  srand48_r(0, &alea_buffer);
}

float uniform_random_number() {
  double res = 0.0; 
  drand48_r(&alea_buffer,&res);
  return res;
}


/*
 * notre gettimeofday()
 */
double my_gettimeofday(){
  struct timeval tmp_time;
  gettimeofday(&tmp_time, NULL);
  return tmp_time.tv_sec + (tmp_time.tv_usec * 1.0e-6L);
}

/*
 * kernel
 */ 
__global__ void neutron_gpu(int n,int* r,int* t,int* b, float* absorbed,float c, float c_c, float c_s, float h, int num,int nb_block){
  // distance parcourue par le neutron avant la collision
  float L;
  // direction du neutron (0 <= d <= PI)
  float d;
  // variable aléatoire uniforme
  float u;
  // position de la particule (0 <= x <= h)
  float x;
  //(n,r,t,b,absorbed,c,c_c,c_s,L,h,d,x,u)
  int j, old, next_ind;
  unsigned int seed;
  hiprandState state;
  
  next_ind = (num + 1) * blockDim.x * nb_block + threadIdx.x + blockIdx.x * blockDim.x;
  j = num * blockDim.x * nb_block + threadIdx.x + blockIdx.x * blockDim.x;
  seed = j;
  hiprand_init(seed, 0, 0, &state); 
  if((threadIdx.x+1)%1024==0)
	printf("threadcpu=%d && threadgpu=%d\n",num,j); 
  if(j < next_ind && j < n){ 
    d = 0.0;
    x = 0.0;

    while (1) {
      u = hiprand_uniform(&state);	
      L = -(1 / c) * log(u);
      x = x + L * cos(d);
      if (x < 0) {
	atomicAdd(r, 1);
	break;
      } else if (x >= h) {
	atomicAdd(t, 1);
	break;
      } else if ((u = hiprand_uniform(&state)) < c_c / c) {
	old = atomicAdd(b, 1);
	absorbed[old] = x;					
	break;
      } else {
	u = hiprand_uniform(&state);
	d = u * M_PI;
      }
    }
  }
}

/*
 * main()
 */
int main(int argc, char *argv[]) {
  // La distance moyenne entre les interactions neutron/atome est 1/c. 
  // c_c et c_s sont les composantes absorbantes et diffusantes de c. 
  float c, c_c, c_s;
  // épaisseur de la plaque
  float h;
  // nombre d'échantillons
  int n;
  // nombre de neutrons refléchis, absorbés et transmis
  int r, b, t;
  // chronometrage
  double start, finish;
  //int i, j = 0; // compteurs 
  int j,i;
  float* absorbed;
  float* g_absorbed;
  int *gpu_r, *gpu_t, *gpu_b;
  int nb_thread_cpu, NB_BLOCK;

  if( argc == 1)
    fprintf( stderr, "%s\n", info);

  // valeurs par defaut
  h = 1.0;
  n = 500000000;
  c_c = 0.5;
  c_s = 0.5;

  // recuperation des parametres
  if (argc > 1)
    h = atof(argv[1]);
  if (argc > 2)
    n = atoi(argv[2]);
  if (argc > 3)
    c_c = atof(argv[3]);
  if (argc > 4)
    c_s = atof(argv[4]);
  r = b = t = 0;
  c = c_c + c_s;
	
  // affichage des parametres pour verificatrion
  printf("Épaisseur de la plaque : %4.g\n", h);
  printf("Nombre d'échantillons  : %d\n", n);
  printf("C_c : %g\n", c_c);
  printf("C_s : %g\n", c_s);

  
  hipMalloc((void**)&g_absorbed, n*sizeof(float));
  hipMalloc((void**)&gpu_b, sizeof(int));
  hipMalloc((void**)&gpu_r, sizeof(int));
  hipMalloc((void**)&gpu_t, sizeof(int));
  
  hipMemcpy(gpu_r, &r, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(gpu_t, &t, sizeof(int), hipMemcpyHostToDevice);  
  hipMemcpy(gpu_b, &b, sizeof(int), hipMemcpyHostToDevice);
  
  absorbed = (float *) calloc(n, sizeof(float));
  hipMemcpy(g_absorbed, absorbed,n*sizeof(float), hipMemcpyHostToDevice);
  
  // debut du chronometrage
  start = my_gettimeofday();

  //Partie à parraléliser
  //echantillon par thread
  nb_thread_cpu = omp_get_max_threads();
  NB_BLOCK =ceil(n/(float)(THREADS_PER_BLOCK * nb_thread_cpu));
  printf("NB_BLOCK=%d && max_thread=%d\n",NB_BLOCK,nb_thread_cpu);
  #pragma omp parallel for
  for (i = 0; i < nb_thread_cpu; i++){
    neutron_gpu<<<NB_BLOCK,THREADS_PER_BLOCK>>>(n, gpu_r, gpu_t, gpu_b, g_absorbed, c, c_c, c_s, h,omp_get_thread_num(),NB_BLOCK);
  }
  
  hipDeviceSynchronize();
  // fin du chronometrage  
  finish = my_gettimeofday();
  printf("num thread = %d \n",omp_get_thread_num());
  hipMemcpy(absorbed, g_absorbed,n*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(&r, gpu_r, sizeof(int), hipMemcpyDeviceToHost);  
  hipMemcpy(&t, gpu_t, sizeof(int), hipMemcpyDeviceToHost);  
  hipMemcpy(&b, gpu_b, sizeof(int), hipMemcpyDeviceToHost);
  
  printf("\nPourcentage des neutrons refléchis : %4.2g\n", (float) r / (float) n);
  printf("Pourcentage des neutrons absorbés : %4.2g\n", (float) b / (float) n);
  printf("Pourcentage des neutrons transmis : %4.2g\n", (float) t / (float) n);
  printf("num thread = %d r=%d b=%d t=%d\n",omp_get_thread_num(),r,b,t);
  printf("\nTemps total de calcul: %.8g sec\n", finish - start);
  printf("Millions de neutrons /s: %.2g\n", (double) n / ((finish - start)*1e6));

  // ouverture du fichier pour ecrire les positions des neutrons absorbés
  FILE *f_handle = fopen(OUTPUT_FILE, "w");
  if (!f_handle) {
    fprintf(stderr, "Cannot open " OUTPUT_FILE "\n");
    exit(EXIT_FAILURE);
  }
  //float *new_end = thrust::remove_if(absorbed, absorbed+n, is_not_zero());
  for (j = 0; j < b; j++){
    fprintf(f_handle, "%f\n", absorbed[j]);
  }

  fclose(f_handle);
  printf("Result written in " OUTPUT_FILE "\n"); 
  hipFree(g_absorbed);
  hipFree(gpu_r);
  hipFree(gpu_t);
  hipFree(gpu_b);
	
  free(absorbed);

  return EXIT_SUCCESS;
}
