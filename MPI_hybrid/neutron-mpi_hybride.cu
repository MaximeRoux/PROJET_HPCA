#include "hip/hip_runtime.h"
/*
 * Université Pierre et Marie Curie
 * Calcul de transport de neutrons
 * Version séquentielle
 */
//nvcc -o exec neutron-par.cu -O3 --generate-code arch =compute_35, code=sm_35 && ./exec
//nvcc -o exec neutron-par.cu -O3 --generate-code arch=compute_35,code=sm_35 && ./exec
#include <unistd.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>
#include<hiprand/hiprand_kernel.h>
#include <thrust/remove.h>

#include<mpi.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <omp.h>

#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#define THREADS_PER_BLOCK 1024
#define OUTPUT_FILE "/tmp/absorbed.dat"

char info[] = "\
Usage:\n\
    neutron-mpi_hybride H Nb C_c C_s\n\
\n\
    H  : épaisseur de la plaque\n\
    Nb : nombre d'échantillons\n\
    C_c: composante absorbante\n\
    C_s: componente diffusante\n\
\n\
Exemple d'execution : \n\
    neutron-seq 1.0 500000000 0.5 0.5\n\
";

/*
 * générateur uniforme de nombres aléatoires dans l'intervalle [0,1)
 */
struct drand48_data alea_buffer;


struct is_not_zero
{
  __host__ __device__
  bool operator()(float x)
  {
    return  x == 0;
  }
};


struct is_even
{
  __host__ __device__
  bool operator()(const int x)
  {
    return (x % 2) == 0;
  }
};


void init_uniform_random_number() {
  srand48_r(0, &alea_buffer);
}

float uniform_random_number(struct drand48_data* r) {
  double res = 0.0;
  drand48_r(r,&res);
  return res;
}

/*
 * notre gettimeofday()
 */
double my_gettimeofday(){
  struct timeval tmp_time;
  gettimeofday(&tmp_time, NULL);
  return tmp_time.tv_sec + (tmp_time.tv_usec * 1.0e-6L);
}

/*
 * kernel
 */
__global__ void neutron_gpu(int n,int* r,int* t,int* b, float* absorbed,float c, float c_c, float c_s, float h, int nb_proc){
    // distance parcourue par le neutron avant la collision
    float L;
    // direction du neutron (0 <= d <= PI)
    float d;
    // variable aléatoire uniforme
    float u;
    // position de la particule (0 <= x <= h)
    float x;
    int j, old;
    unsigned int seed;
    hiprandState state;
  
    j = threadIdx.x+blockIdx.x*blockDim.x;
    seed = j;
  
  
    hiprand_init(seed, 0, 0, &state); 

    if(j<(n/nb_proc)){ 
			d = 0.0;
			x = 0.0;

			while (1) {
					u = hiprand_uniform(&state);
		
					L = -(1 / c) * log(u);
					x = x + L * cos(d);
					if (x < 0) {
						atomicAdd(r, 1);
						break;
					} else if (x >= h) {
						atomicAdd(t, 1);
						break;
					} 
	
					else if ((u = hiprand_uniform(&state)) < c_c / c) {
						old = atomicAdd(b, 1);
						absorbed[old] = x;					
						break;
					} else {
						u = hiprand_uniform(&state);
						d = u * M_PI;
					}
			}

    }
} 

/*
 * main()
 */
int main(int argc, char *argv[]) {
  // La distance moyenne entre les interactions neutron/atome est 1/c.
  // c_c et c_s sont les composantes absorbantes et diffusantes de c.
  float c, c_c, c_s;
  // épaisseur de la plaque
  float h;
  // nombre d'échantillons
  int n;
  // nombre de neutrons refléchis, absorbés et transmis
  int r, b, t;
  // chronometrage
  double start, finish;
  //int i, j = 0; // compteurs
  int i;
  int j=0;
  float* absorbed;
  float* c_absorbed;
  float* g_absorbed;
  int *gpu_r, *gpu_t, *gpu_b;
  // int epth;
  int NB_BLOCK;
  int n_cpu, n_gpu;
  int r_aux, b_aux, t_aux;
 
 	int global_b,global_r,global_t;
 	int global_b_aux,global_r_aux,global_t_aux;
  unsigned int seed;
  float L;
  float d;
  float u;
  float x;
  float pn_cpu=0.1;
  struct drand48_data test;
  double finish_cpu,finish_gpu,start_cpu,start_gpu;
  int tmp=0;
  int my_rank, nb_proc;

	MPI_Init(&argc,&argv);
	MPI_Comm_size(MPI_COMM_WORLD,&nb_proc);
  MPI_Comm_rank(MPI_COMM_WORLD,&my_rank);
 	
  if( argc == 1)
    fprintf( stderr, "%s\n", info);

  // valeurs par defaut
  h = 1.0;
  n = 500000000;
  c_c = 0.5;
  c_s = 0.5;
 
  // recuperation des parametres
  if (argc > 1)
    h = atof(argv[1]);
  if (argc > 2)
    n = atoi(argv[2]);
  if (argc > 3)
    c_c = atof(argv[3]);
  if (argc > 4)
    c_s = atof(argv[4]);
  if (argc > 5)
    pn_cpu = atof(argv[5]);
  r = b = t = 0;
  c = c_c + c_s;

  n_cpu = (int)(pn_cpu * n);
  n_gpu = n - n_cpu;
 	printf("n_cpu=%d n_gpu=%d n=%d\n",n_cpu,n_gpu,n_cpu+n_gpu);
  // affichage des parametres pour verificatrion
  printf("Épaisseur de la plaque : %4.g\n", h);
  printf("Nombre d'échantillons  : %d\n", n);
  printf("C_c : %g\n", c_c);
  printf("C_s : %g\n", c_s);

 
  
  absorbed = (float *) calloc(n_gpu/nb_proc, sizeof(float));
 	c_absorbed = (float *) calloc(n_cpu/nb_proc, sizeof(float));
  // debut du chronometrage
 
  //Partie à parraléliser
  //echantillon par thread
  //epth = omp_get_max_threads();
  NB_BLOCK = (n_gpu+THREADS_PER_BLOCK*nb_proc-1)/(THREADS_PER_BLOCK*nb_proc);

  MPI_Barrier(MPI_COMM_WORLD);
  start = my_gettimeofday();
	
  #pragma omp parallel private (seed, x, L, u, d, tmp, test) shared(r,t,b)
  {
  	seed=omp_get_thread_num();
  	srand48_r(seed,&test);
  	
    #pragma omp master
    {    
      start_gpu = my_gettimeofday();
      hipMalloc((void**)&g_absorbed, (n_gpu/nb_proc)*sizeof(float));
			hipMalloc((void**)&gpu_b, sizeof(int));
			hipMalloc((void**)&gpu_r, sizeof(int));
			hipMalloc((void**)&gpu_t, sizeof(int));
      hipMemcpy(gpu_r, &r, sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(gpu_t, &t, sizeof(int), hipMemcpyHostToDevice); 
			hipMemcpy(gpu_b, &b, sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(g_absorbed, absorbed,(n_gpu/nb_proc)*sizeof(float), hipMemcpyHostToDevice);
			
      neutron_gpu<<<NB_BLOCK,THREADS_PER_BLOCK>>>(n_gpu, gpu_r, gpu_t, gpu_b, g_absorbed, c,
                          c_c, c_s, h,nb_proc); 
      hipDeviceSynchronize();

      hipMemcpy(absorbed, g_absorbed,(n_gpu/nb_proc)*sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(&r_aux, gpu_r, sizeof(int), hipMemcpyDeviceToHost); 
			hipMemcpy(&t_aux, gpu_t, sizeof(int), hipMemcpyDeviceToHost); 
			hipMemcpy(&b_aux, gpu_b, sizeof(int), hipMemcpyDeviceToHost);
      finish_gpu = my_gettimeofday();
      printf("\nTemps total de calcul GPU: %.8g sec\n", finish_gpu - start_gpu);
    }
		

    
   	start_cpu = omp_get_wtime();
    #pragma omp for reduction(+:r) reduction(+:b) reduction(+:t)
    for (i = 0; i < (n_cpu)/nb_proc; i++) {
      d = 0.0;
      x = 0.0;
      while (1) {
   
				u = uniform_random_number(&test);
				L = -(1 / c) * log(u);
				x = x + L * cos(d);
				if (x < 0) {
				  r++;
				  break;
				}else if (x >= h) {
				  t++;
				  break;
				} else if ((u = uniform_random_number(&test)) < c_c / c) {
				  b++;
				  // fonctionne si on enleve tmp de private
				  // on laisse les deux ~9sec et ~53n/s
				  // sans le atomic ~8sec et ~61n/s
				  // sans le private tmp ~9sec et ~53n/s
				  // 8 threads  
				  #pragma omp atomic capture
				  tmp=j++;
				  /*if(tmp>=n_cpu)
				  	printf("tmp=%d\n",tmp);*/
				  c_absorbed[tmp] = x;
				  break;
				} else {
				  u = uniform_random_number(&test);
				  d = u * M_PI;
				}
			}
    }
    finish_cpu = omp_get_wtime();
  }
  
 	MPI_Reduce(&b, &global_b, 1, MPI_INT, MPI_SUM, 0,MPI_COMM_WORLD);
  MPI_Reduce(&t, &global_t, 1, MPI_INT, MPI_SUM, 0,MPI_COMM_WORLD);
  MPI_Reduce(&r, &global_r, 1, MPI_INT, MPI_SUM, 0,MPI_COMM_WORLD);
  
  MPI_Reduce(&b_aux, &global_b_aux, 1, MPI_INT, MPI_SUM, 0,MPI_COMM_WORLD);
  MPI_Reduce(&t_aux, &global_t_aux, 1, MPI_INT, MPI_SUM, 0,MPI_COMM_WORLD);
  MPI_Reduce(&r_aux, &global_r_aux, 1, MPI_INT, MPI_SUM, 0,MPI_COMM_WORLD);
  global_r += global_r_aux;
  global_b += global_b_aux;
  global_t += global_t_aux;
  MPI_Barrier(MPI_COMM_WORLD);
  finish = my_gettimeofday();
  if(my_rank==0){
  	printf("Nombre neutrons refléchis : %d\n",global_r);
  	printf("Nombre neutrons absorbés : %d\n",global_b);
  	printf("Nombre neutrons transmis : %d\n",global_t);
		printf("\nPourcentage des neutrons refléchis : %4.2g\n", (float) global_r / (float) n);
		printf("Pourcentage des neutrons absorbés : %4.2g\n", (float) global_b / (float) n);
		printf("Pourcentage des neutrons transmis : %4.2g\n", (float) global_t / (float) n);
		
		printf("\nTemps total de calcul C: %.8g sec\n", finish_cpu - start_cpu);
		printf("\nTemps total de calcul: %.8g sec\n", finish - start);
		printf("Millions de neutrons /s: %.2g\n", (double) n / ((finish - start)*1e6));

	}
  // ouverture du fichier pour ecrire les positions des neutrons absorbés
  /*
  FILE *f_handle = fopen(OUTPUT_FILE, "w");
  if (!f_handle) {
    fprintf(stderr, "Cannot open " OUTPUT_FILE "\n");
    exit(EXIT_FAILURE);
  }
  
  //float *new_end = thrust::remove_if(absorbed, absorbed+n, is_not_zero());
  for (j = 0; j < b_aux; j++){
    fprintf(f_handle, "%f\n", absorbed[j]);
  }
	for (j = b_aux; j < b; j++){
    fprintf(f_handle, "%f\n", c_absorbed[j-b_aux]);
  }
  fclose(f_handle);
  printf("Result written in " OUTPUT_FILE "\n");
  */
  hipFree(g_absorbed);
  hipFree(gpu_r);
  hipFree(gpu_t);
  hipFree(gpu_b);
   
  free(absorbed);

  return EXIT_SUCCESS;
}
